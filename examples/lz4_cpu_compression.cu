/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "BatchData.h"

#include "lz4.h"
#include "lz4hc.h"
#include "nvcomp/lz4.h"

#define CHECK_NVCOMP_STATUS(status)                                            \
  if ((status) != nvcompSuccess)                                               \
    throw std::runtime_error("Failed to decompress data");

// Benchmark performance from the binary data file fname
static void run_example(const std::vector<std::vector<char>>& data)
{
  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;

  // build up input batch on CPU
  BatchDataCPU input_data_cpu(data, chunk_size);
  std::cout << "chunks: " << input_data_cpu.size() << std::endl;

  // compression

  // Allocate and prepare output/compressed batch
  BatchDataCPU compress_data_cpu(
      LZ4_compressBound(chunk_size), input_data_cpu.size());

  // loop over chunks on the CPU, compressing each one
  for (size_t i = 0; i < input_data_cpu.size(); ++i) {
    // could use LZ4_compress_default or LZ4_compress_fast instead
    const int size = LZ4_compress_HC(
        static_cast<const char*>(input_data_cpu.ptrs()[i]),
        static_cast<char*>(compress_data_cpu.ptrs()[i]),
        input_data_cpu.sizes()[i],
        compress_data_cpu.sizes()[i],
        12);
    if (size == 0) {
      throw std::runtime_error(
          "LZ4 CPU failed to compress chunk " + std::to_string(i) + ".");
    }

    // set the actual compressed size
    compress_data_cpu.sizes()[i] = size;
  }

  // compute compression ratio
  size_t* compressed_sizes_host = compress_data_cpu.sizes();
  size_t comp_bytes = 0;
  for (size_t i = 0; i < compress_data_cpu.size(); ++i)
    comp_bytes += compressed_sizes_host[i];

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;

  // Copy compressed data to GPU
  BatchData compress_data(compress_data_cpu, true);

  // Allocate and build up decompression batch on GPU
  BatchData decomp_data(input_data_cpu, false);

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // CUDA events to measure decompression time
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  // lz4 GPU decompression
  size_t decomp_temp_bytes;
  nvcompError_t status = nvcompBatchedLZ4DecompressGetTempSize(
      compress_data.size(), chunk_size, &decomp_temp_bytes);
  CHECK_NVCOMP_STATUS(status);

  void* d_decomp_temp;
  CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));

  CUDA_CHECK(hipStreamSynchronize(stream));

  // Run decompression
  status = nvcompBatchedLZ4DecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      chunk_size,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      stream);
  CHECK_NVCOMP_STATUS(status);

  // Validate decompressed data against input
  if (!(input_data_cpu == decomp_data))
    throw std::runtime_error("Failed to validate decompressed data");
  else
    std::cout << "decompression validated :)" << std::endl;

  // Re-run decompression to get throughput
  hipEventRecord(start, stream);
  status = nvcompBatchedLZ4DecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      chunk_size,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      stream);
  hipEventRecord(end, stream);
  CHECK_NVCOMP_STATUS(status);

  CUDA_CHECK(hipStreamSynchronize(stream));

  float ms;
  hipEventElapsedTime(&ms, start, end);

  double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
  std::cout << "decompression throughput (GB/s): " << decompression_throughput
            << std::endl;

  hipFree(d_decomp_temp);

  hipEventDestroy(start);
  hipEventDestroy(end);
  hipStreamDestroy(stream);
}
#undef CHECK_NVCOMP_STATUS

std::vector<char> readFile(const std::string& filename)
{
  std::vector<char> buffer(4096);
  std::vector<char> host_data;

  std::ifstream fin(filename, std::ifstream::binary);
  fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);

  size_t num;
  do {
    num = fin.readsome(buffer.data(), buffer.size());
    host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
  } while (num > 0);

  return host_data;
}

std::vector<std::vector<char>>
multi_file(const std::vector<std::string>& filenames)
{
  std::vector<std::vector<char>> split_data;

  for (auto const& filename : filenames) {
    split_data.emplace_back(readFile(filename));
  }

  return split_data;
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names(argc - 1);

  if (argc == 1) {
    std::cerr << "Must specify at least one file." << std::endl;
    return 1;
  }

  // if `-f` is speficieid, assume single file mode
  if (strcmp(argv[1], "-f") == 0) {
    if (argc == 2) {
      std::cerr << "Missing file name following '-f'" << std::endl;
      return 1;
    } else if (argc > 3) {
      std::cerr << "Unknown extra arguments with '-f'." << std::endl;
      return 1;
    }

    file_names = {argv[2]};
  } else {
    // multi-file mode
    for (int i = 1; i < argc; ++i) {
      file_names[i - 1] = argv[i];
    }
  }

  auto data = multi_file(file_names);

  run_example(data);

  return 0;
}
